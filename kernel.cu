#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include "chaos_game.h"
#include "hiprand/hiprand_kernel.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void setup_states(hiprandState * states, int seed) {
	int id = threadIdx.x + blockIdx.x * blockDim.x; 
	hiprand_init(seed, id, 0, &states[id]);
}

// dev_count points is repeated once per thread block.
// Each thread block should have max 32 threads.
// counts must have been allocated using cuda-allocate in global memory.
__global__ void dev_count_points(int num_points, int num_vertices, float * vertices, int num_iterations, int * counts, int length, hiprandState * states) {
	extern __shared__ float dev_vertices[];
	if (threadIdx.x == 0) {
		for (int i = 0; i < num_vertices * 2; i++) {
			dev_vertices[i] = vertices[i];
		}
	}
	__syncthreads();
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState state = states[id];
	float radius = ((float)length) / 2;
	for (int p = 0; p < num_points; p++) {
		float current_x = radius;
		float current_y = radius;
		for (int i = 0; i < num_iterations; i++) {
			int r = (int)truncf(((float)num_vertices - 0.000001) * hiprand_uniform(&state));
			current_x += (dev_vertices[2 * r] - current_x) / 2;
			current_y += (dev_vertices[2 * r + 1] - current_y) / 2;
		}
		counts[length * (int)truncf(current_y) + (int)truncf(current_x)] += 1;
	}
}

int main()
{
	/*
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	*/

	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();

	int num_blocks = 28;
	int num_threads = 128;
	int length = 4000;
	int num_vertices = 9;
	int num_points = 400000000;
	int num_iterations = 25;
	
	float * vertices = make_vertices(length, num_vertices);
	int * counts = new int[length * length]();
	float * dev_vertices;
	int * dev_counts;
	hiprandState * dev_states;
	hipMalloc((void**)&dev_vertices, 2 * num_vertices * sizeof(float));
	hipMalloc((void**)&dev_states, num_blocks * num_threads * sizeof(hiprandState));
	hipMalloc((void**)&dev_counts, length * length * sizeof(int));

	hipMemset(dev_counts, 0, length * length * sizeof(int));
	hipMemcpy(dev_vertices, vertices, 2 * num_vertices * sizeof(float), hipMemcpyHostToDevice);

	setup_states <<<num_blocks, num_threads >>> (dev_states, seed);

	dev_count_points <<<num_blocks, num_threads, 2 * num_vertices * sizeof(float) >>> (num_points, num_vertices, dev_vertices, num_iterations, dev_counts, length, dev_states);

	hipError_t status = hipMemcpy(counts, dev_counts, length * length * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_vertices);
	hipFree(dev_states);
	hipFree(dev_counts);

	save_fractal(length, counts, "test.png");
	
	delete[] counts;

    return 0;
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
